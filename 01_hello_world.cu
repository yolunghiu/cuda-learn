
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__告诉编译器这个是个可以在设备上执行的核函数
__global__ void hello_world(void)
{
    printf("GPU: Hello world!\n");
}

int main(int argc, char** argv)
{
    printf("CPU: Hello world!\n");
    hello_world<<<1, 3>>>();

    // 等GPU执行完了，再退出主机线程
    // if no this line ,it can not output hello world from gpu
    hipDeviceReset();
    return 0;
}