#include <hip/hip_runtime.h>


#include <iostream>

using namespace std;

// 二：线程执行代码
// dim3 grid(1, 1, 1), block(length, 1, 1);
__global__ void vector_add1(float* vec1, float* vec2, float* vecres, int length)
{
    int tid = threadIdx.x;
    if (tid < length)
    {
        vecres[tid] = vec1[tid] + vec2[tid];
    }
}

// dim3 grid(16, 1, 1), block(1, 1, 1);
__global__ void vector_add2(float* vec1, float* vec2, float* vecres, int length)
{
    int tid = blockIdx.x;
    if (tid < length)
    {
        vecres[tid] = vec1[tid] + vec2[tid];
    }
}

// dim3 grid(1, 1, 1), block(4, 4, 1);
__global__ void vector_add3(float* vec1, float* vec2, float* vecres, int length)
{
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    if (tid < length)
    {
        vecres[tid] = vec1[tid] + vec2[tid];
    }
}

// dim3 grid(4, 1, 1), block(4, 1, 1);
__global__ void vector_add4(float* vec1, float* vec2, float* vecres, int length)
{
    int tid = blockIdx.x * gridDim.x + threadIdx.x;
    if (tid < length)
    {
        vecres[tid] = vec1[tid] + vec2[tid];
    }
}

// dim3 grid(2, 2, 1), block(2, 2, 1);
__global__ void vector_add5(float* vec1, float* vec2, float* vecres, int length)
{
    int tid =
        (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) +
        threadIdx.y * blockDim.y + threadIdx.x;
    if (tid < length)
    {
        vecres[tid] = vec1[tid] + vec2[tid];
    }
}

int main()
{
    const int length = 16;                  // 数组长度为16
    float a[length], b[length], c[length];  // host中的数组
    for (int i = 0; i < length; i++)
    {  // 初始赋值
        a[i] = b[i] = i;
    }
    float *a_device, *b_device, *c_device;  // device中的数组

    hipMalloc((void**)&a_device, length * sizeof(float));  // 分配内存
    hipMalloc((void**)&b_device, length * sizeof(float));
    hipMalloc((void**)&c_device, length * sizeof(float));

    hipMemcpy(a_device,
               a,
               length * sizeof(float),
               hipMemcpyHostToDevice);  // 将host数组的值拷贝给device数组
    hipMemcpy(b_device, b, length * sizeof(float), hipMemcpyHostToDevice);

    // 一：参数配置
    //    dim3 grid(1, 1, 1), block(length, 1, 1);  // 设置参数
    //    dim3 grid(length, 1, 1), block(1, 1, 1);  // 设置参数
    //    dim3 grid(1, 1, 1), block(4, 4, 1);
    //    dim3 grid(4, 1, 1), block(4, 1, 1);
    dim3 grid(2, 2, 1), block(2, 2, 1);
    vector_add5<<<grid, block>>>(
        a_device, b_device, c_device, length);  // 启动kernel

    hipMemcpy(c,
               c_device,
               length * sizeof(float),
               hipMemcpyDeviceToHost);  // 将结果拷贝到host

    for (int i = 0; i < length; i++)
    {  // 打印出来方便观察
        cout << c[i] << " ";
    }
    
    return 0;
}
